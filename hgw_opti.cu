#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <vector>
#include <string>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define max(a,b) a>b?a:b
#define min(a,b) a<b?a:b


__global__ void print_cuda(size_t* data, int height, int width) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x >= width || y >= height) {
		return;
	}
	printf("%i, %i --> %lu\n", x, y, data[x+y*width]);
}


__global__ void compute_vHGW(size_t* data_read, size_t* data_write, int height, int width, size_t* g, size_t* h, size_t k, bool is_dilatation) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	auto m = width;
	auto psa = (k - (m - 1) % k) - 1;

	if (index >= width)
		return;

	size_t* curr_line = data_read+index*width;
	size_t* g_line = g+index*width;
	size_t* h_line = h+index*width;
	size_t* v_line = data_write+index*width;

	auto compare = is_dilatation ? max : min
	
	for (int x = 0; x < m; x++) {
	  // Compute G
      g_line[x] = (x % k) == 0 ? curr_line[x] : compare(g_line[x - 1], curr_line[x], is_dilatation);
      // Compute H
      int x_rev = m - x - 1;
      if (x_rev == m-1) {
      	h_line[x_rev] = curr_line[x_rev];
      } else {
      	h_line[x] = (x_rev + 1) % k == 0 ? curr_line[x_rev] : compare(h_line[x_rev + 1], curr_line[x_rev], is_dilatation);
      }
	}

    // Compute new line 
    for (size_t x = 0; x < m; x++)
    {
      if (2*x < k)
        v_line[x] = g_line[x + k/2];
      else if (x + k/2 >= m)
        v_line[x] = x + k/2 < m + psa ? compare(g_line[m - 1], h_line[x - k/2], is_dilatation) : h_line[x - k/2];
      else
        v_line[x] = compare(g_line[x + k/2], h_line[x - k/2], is_dilatation);
    }

}

void cuda_vHGW(size_t* data_host, int height, int width, size_t k, bool	is_dilatation) {
	size_t* data_read;
	size_t* data_write;
	size_t* h;
	size_t* g;

	// Allocate device memory 
	hipMalloc(&data_read, sizeof(size_t) * height * width);
	hipMalloc(&data_write, sizeof(size_t) * height * width);
	hipMalloc(&g, sizeof(size_t) * height * width);
	hipMalloc(&h, sizeof(size_t) * height * width);

	// Transfer data from host to device memory
	hipMemcpy(data_read, data_host, sizeof(size_t) * width * height, hipMemcpyHostToDevice);

	//int bsize = 1;
	//int ww = std::ceil((float)width / bsize);
	//int hh = std::ceil((float)height / bsize);

	// Executing kernel 
	//dim3 dimBlock(bsize, bsize);
	//dim3 dimGrid(w, h);

	printf("BEFORE\n");
	compute_vHGW<<<height, 1>>>(data_read, data_write, height, width, g, h, k, is_dilatation);
	hipDeviceSynchronize();
	printf("AFTER\n");

	// Transfer data back to host memory
	hipMemcpy(data_host, data_write, sizeof(size_t) * width * height, hipMemcpyDeviceToHost);

	// Deallocate device memory
    hipFree(data_read);
    hipFree(data_write);
    hipFree(h);
    hipFree(g);
}


int main() {
	size_t* data;
	int height = 10;
	int width  = 10;
	bool is_dilatation = true;

	data = (size_t*)malloc(sizeof(size_t) * height*width);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			data[j + i * width] = (i * width) +j;
		}
	}


	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			printf("%lu ", data[j+i*width]);
		}
		printf("\n");
	}

	size_t k = 3;

	cuda_vHGW(data, height, width, k, is_dilatation);


	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			printf("%lu ", data[j+i*width]);
		}
		printf("\n");
	}

	return 0;

}